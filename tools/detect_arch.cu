#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int device = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    printf("sm_%d%d", prop.major, prop.minor);
    return 0;
}
